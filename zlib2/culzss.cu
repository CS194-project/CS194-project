#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <sys/time.h>
#include <errno.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <algorithm>
#include "deflate.h"
#include "culzss.h"

void
checkCPUError (const char *msg)
{
    const int BUF_SIZE = 1024;
    char buf[BUF_SIZE];
    if (errno != 0 && errno != EEXIST)
    {
        char *errmsg = strerror_r (errno, buf, 1024);
        fprintf (stderr, "CPU error: %s: %s.\n", msg, errmsg);
        exit (1);
    }
}

void
checkCudaError (const char *msg)
{
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err)
    {
        fprintf (stderr, "Cuda error: %s: %s.\n", msg,
                 hipGetErrorString (err));
        exit (EXIT_FAILURE);
    }
}

void
culzss_init (deflate_state *s)
{
  /*
    for (int i = 0; i < CULZSS_CUDA_NUM_STREAMS; i++)
    {
        hipStreamCreate (&(s->streams[i]));
        checkCudaError ("Cuda Create stream.");
        }*/

    /* Need initial WINDOW_SIZE bytes as initial window. EXTRA BUF to avoid out
     * of bound memory error. */
    // hipHostAlloc (&s->host_in,
    //               sizeof (*s->host_in) * (CULZSS_MAX_PROCESS_SIZE + CULZSS_WINDOW_SIZE +
    //                                  CULZSS_EXTRA_BUF),
    //               hipHostMallocDefault);
//checkCudaError ("Allocate host_in");
//
    s->host_in = (unsigned char*)malloc(
                   sizeof (*s->host_in) * (CULZSS_MAX_PROCESS_SIZE + CULZSS_WINDOW_SIZE +
                                               CULZSS_EXTRA_BUF));

    hipMalloc (&s->device_in,
                sizeof (*s->device_in) * (CULZSS_MAX_PROCESS_SIZE + CULZSS_WINDOW_SIZE +
                                          CULZSS_EXTRA_BUF));
    checkCudaError ("Allocate device_in");

    s->host_encode = (culzss_encoded_string_t*)malloc(
                   sizeof (*s->host_encode) * (CULZSS_MAX_PROCESS_SIZE + CULZSS_WINDOW_SIZE +
                                               CULZSS_EXTRA_BUF));
    checkCudaError ("Allocate host_encode");
    hipMalloc (&s->device_encode,
                sizeof (*s->device_encode) * (CULZSS_MAX_PROCESS_SIZE + CULZSS_WINDOW_SIZE +
                                              CULZSS_EXTRA_BUF));
    checkCudaError ("Allocate device_encode");
}


void
culzss_destroy (deflate_state *s)
{
    /* Clean up */
  /*
    for (int i = 0; i < CULZSS_CUDA_NUM_STREAMS; i++)
    {
        if (s->streams[i] != NULL)
            hipStreamDestroy (s->streams[i]);
        s->streams[i] = NULL;
        }*/

    if (s->device_in != NULL)
        hipFree (s->device_in);
    s->device_in = NULL;

    if (s->device_encode != NULL)
        hipFree (s->device_encode);
    s->device_encode = NULL;

        if (s->host_in != NULL)
        hipHostFree (s->host_in);
     s->host_in = NULL;

    if (s->host_encode != NULL)
        free (s->host_encode);
    s->host_encode = NULL;
}

__global__ void
lzss_kernel (const unsigned char *__restrict__ in_g,
             culzss_encoded_string_t * __restrict__ encode, int grid_size,
             int is_firstblock)
{
    __shared__ unsigned char in[CULZSS_WINDOW_SIZE * 2];	/* Note that WINDOW_SIZE must be a
                                                               multiple of blockDimension(1024).
                                                               First half are window and second half are
                                                               lookahead. */
    __shared__ unsigned short hashtable[CULZSS_HASH_SIZE];
    const int CULZSS_STEP_SIZE = 64 * 1024;
    /* We do 64KB every step. (Actual 64KB - 4KB (window size) because of
       overlapping).
       This is because we use unsigned short as the
       type of hashtable to maximize the number of indexes in hashtable.
       It can store the indexes to the maximum to 65535. */

    while (grid_size > 0)
    {
        int block_size = grid_size - (CULZSS_STEP_SIZE * blockIdx.x -
                                      CULZSS_WINDOW_SIZE * blockIdx.x);
        if (block_size > CULZSS_STEP_SIZE)
            block_size = CULZSS_STEP_SIZE;
        encode += (CULZSS_STEP_SIZE - CULZSS_WINDOW_SIZE) * blockIdx.x;
        in_g += (CULZSS_STEP_SIZE - CULZSS_WINDOW_SIZE) * blockIdx.x;

        /* initialize shared memory. */
        /* Note that in_g needs to have some extra space at the end to avoid invalid
           memory access */
        for (int i = threadIdx.x * 4; i < CULZSS_WINDOW_SIZE * 2; i += blockDim.x * 4)
        {
            *((int *) (in + i)) = *((int *) (in_g + i));
        }
        __syncthreads ();

        /* Compute hash of initial sliding window. */
        for (int ii = threadIdx.x * 4; ii < CULZSS_WINDOW_SIZE; ii += blockDim.x * 4)
        {
            for (int i = ii; i < ii + 4; i++)
            {
                int hash = 0;
                hash = (((hash) << CULZSS_HASH_SHIFT) ^ (in[i]));
                hash = (((hash) << CULZSS_HASH_SHIFT) ^ (in[i + 1]));
                hash = (((hash) << CULZSS_HASH_SHIFT) ^ (in[i + 2]));
                hash = hash & CULZSS_HASH_MASK;
                hashtable[hash] = i;
            }
        }
        __syncthreads ();

        int hash0 = 0, index0 = threadIdx.x;
        int hash1 = 0, index1 = threadIdx.x;
        int hash2 = 0, index2 = threadIdx.x;
        int hash3 = 0, index3 = threadIdx.x;
        for (int uncodedHead = CULZSS_WINDOW_SIZE; uncodedHead < block_size;
             uncodedHead += CULZSS_WINDOW_SIZE)
        {
            int end = CULZSS_WINDOW_SIZE + min (block_size - uncodedHead, CULZSS_WINDOW_SIZE);

            for (int i = threadIdx.x + CULZSS_WINDOW_SIZE; i < 2 * CULZSS_WINDOW_SIZE;
                 i += blockDim.x)
            {
                unsigned int hash = 0;
                unsigned char char0 = in[i];
                unsigned char char1 = 0;
                unsigned char char2 = 0;
                unsigned char char3 = 0;
                unsigned char char4 = 0;
                unsigned char char5 = 0;
                culzss_encoded_string_t match_data;
                int match_len;
                unsigned int prev;
                match_data.len = 0;
                match_data.dist = 0;

                /* Load characters */
                if (i < end - MAX_MATCH + 1)
                {
                    char0 = in[i];
                    char1 = in[i + 1];
                    char2 = in[i + 2];
                    char3 = in[i + 3];
                    char4 = in[i + 4];
                    char5 = in[i + 5];
                    /* Compute hash key of 3 characters. */
                    hash = (((hash) << CULZSS_HASH_SHIFT) ^ (char0));
                    hash = (((hash) << CULZSS_HASH_SHIFT) ^ (char1));
                    hash = (((hash) << CULZSS_HASH_SHIFT) ^ (char2));
                    hash = hash & CULZSS_HASH_MASK;
                }

                /* Check most recent hash. */
                __syncthreads ();
                prev = hashtable[hash];
                prev = prev + CULZSS_WINDOW_SIZE - uncodedHead;
                match_len = 0;

                if (prev > 0
                    && prev < i - CULZSS_MAX_MATCH + 1
                    && i - prev <= CULZSS_WINDOW_SIZE && i < end - CULZSS_MAX_MATCH + 1)
                {
                    int temp = 1;	/* Used to check match. */
                    temp *= (in[prev] == char0);
                    match_len += temp;
                    temp *= (in[prev + 1] == char1);
                    match_len += temp;
                    temp *= (in[prev + 2] == char2);
                    match_len += temp;
                    temp *= (in[prev + 3] == char3);
                    match_len += temp;
                    temp *= (in[prev + 4] == char4);
                    match_len += temp;
                    temp *= (in[prev + 5] == char5);
                    match_len += temp;
                }
                if (match_len > match_data.len)
                {
                    match_data.dist = i - prev;
                    match_data.len = match_len;
                }

                /* Check second recent hash. */
                __syncthreads ();
                hashtable[hash1] = index1;
                __syncthreads ();

                prev = hashtable[hash];
                prev = prev + CULZSS_WINDOW_SIZE - uncodedHead;
                match_len = 0;
                if (prev > 0
                    && prev < i - CULZSS_MAX_MATCH + 1
                    && i - prev <= CULZSS_WINDOW_SIZE && i < end - CULZSS_MAX_MATCH + 1)
                {
                    int temp = 1;	/* Used to check match. */
                    temp *= (in[prev] == char0);
                    match_len += temp;
                    temp *= (in[prev + 1] == char1);
                    match_len += temp;
                    temp *= (in[prev + 2] == char2);
                    match_len += temp;
                    temp *= (in[prev + 3] == char3);
                    match_len += temp;
                    temp *= (in[prev + 4] == char4);
                    match_len += temp;
                    temp *= (in[prev + 5] == char5);
                    match_len += temp;
                }
                if (match_len > match_data.len)
                {
                    match_data.dist = i - prev;
                    match_data.len = match_len;
                }


                /* Check third recent hash. */
                __syncthreads ();
                hashtable[hash2] = index2;
                __syncthreads ();

                prev = hashtable[hash];
                prev = prev + CULZSS_WINDOW_SIZE - uncodedHead;
                match_len = 0;
                if (prev > 0
                    && prev < i - CULZSS_MAX_MATCH + 1
                    && i - prev <= CULZSS_WINDOW_SIZE && i < end - CULZSS_MAX_MATCH + 1)
                {
                    int temp = 1;	/* Used to check match. */
                    temp *= (in[prev] == char0);
                    match_len += temp;
                    temp *= (in[prev + 1] == char1);
                    match_len += temp;
                    temp *= (in[prev + 2] == char2);
                    match_len += temp;
                    temp *= (in[prev + 3] == char3);
                    match_len += temp;
                    temp *= (in[prev + 4] == char4);
                    match_len += temp;
                    temp *= (in[prev + 5] == char5);
                    match_len += temp;
                }
                if (match_len > match_data.len)
                {
                    match_data.dist = i - prev;
                    match_data.len = match_len;
                }

                /* Check forth recent hash. */
                __syncthreads ();
                hashtable[hash3] = index3;
                __syncthreads ();

                prev = hashtable[hash];
                prev = prev + CULZSS_WINDOW_SIZE - uncodedHead;
                match_len = 0;
                if (prev > 0
                    && prev < i - CULZSS_MAX_MATCH + 1 && i - prev <= CULZSS_WINDOW_SIZE
                    && i < end - CULZSS_MAX_MATCH + 1)
                {
                    int temp = 1;	/* Used to check match. */
                    temp *= (in[prev] == char0);
                    match_len += temp;
                    temp *= (in[prev + 1] == char1);
                    match_len += temp;
                    temp *= (in[prev + 2] == char2);
                    match_len += temp;
                    temp *= (in[prev + 3] == char3);
                    match_len += temp;
                    temp *= (in[prev + 4] == char4);
                    match_len += temp;
                    temp *= (in[prev + 5] == char5);
                    match_len += temp;
                }
                if (match_len > match_data.len)
                {
                    match_data.dist = i - prev;
                    match_data.len = match_len;
                }

                /* Update recent hash */
                hash3 = hash2;
                index3 = index2;
                hash2 = hash1;
                index2 = index1;
                hash1 = hash0;
                index1 = index0;
                hash0 = hash0;
                index0 = i + uncodedHead - CULZSS_WINDOW_SIZE;

                /* Check current hash. */
                __syncthreads ();
                hashtable[hash] = index0;
                __syncthreads ();

                prev = hashtable[hash];
                prev = prev + CULZSS_WINDOW_SIZE - uncodedHead;
                match_len = 0;
                if (prev > 0
                    && prev < i - CULZSS_MAX_MATCH + 1 && i - prev <= CULZSS_WINDOW_SIZE
                    && i < end - CULZSS_MAX_MATCH + 1)
                {
                    int temp = 1;	/* Used to check match. */
                    temp *= (in[prev] == char0);
                    match_len += temp;
                    temp *= (in[prev + 1] == char1);
                    match_len += temp;
                    temp *= (in[prev + 2] == char2);
                    match_len += temp;
                    temp *= (in[prev + 3] == char3);
                    match_len += temp;
                    temp *= (in[prev + 4] == char4);
                    match_len += temp;
                    temp *= (in[prev + 5] == char5);
                    match_len += temp;
                }

                if (match_len > match_data.len)
                {
                    match_data.dist = i - prev;
                    match_data.len = match_len;
                }

                /* Don't compress first block because initial window can be arbitrary. */
                if ((blockIdx.x == 0 && uncodedHead == CULZSS_WINDOW_SIZE
                     && is_firstblock) || match_data.len < CULZSS_MIN_MATCH)
                {
                    match_data.dist = 0;
                    match_data.len = (unsigned short)char0; /* Store string literal in
                                                               match_data.len. (We know its a literal by match_data.dist
                                                               == 0) */
                }
                /* Done at this position. Store to global memory. */
                if (i < end)
                {
                    encode[i + uncodedHead - CULZSS_WINDOW_SIZE] = match_data;
                }
            }

            __syncthreads ();

            /* Move sliding window. */
            for (int j = threadIdx.x * 4; j < CULZSS_WINDOW_SIZE; j += blockDim.x * 4)
            {
                *((int *) (in + j)) = *((int *) (in + j + CULZSS_WINDOW_SIZE));
            }

            for (int j = threadIdx.x * 4; j < CULZSS_WINDOW_SIZE; j += blockDim.x * 4)
            {
                *((int *) (in + CULZSS_WINDOW_SIZE + j)) =
                    *((int *) (in_g + uncodedHead + CULZSS_WINDOW_SIZE + j));
            }

            __syncthreads ();
        }

        grid_size -= CULZSS_STEP_SIZE - CULZSS_WINDOW_SIZE;
        encode += (CULZSS_STEP_SIZE - CULZSS_WINDOW_SIZE) * gridDim.x;
        in_g += (CULZSS_STEP_SIZE - CULZSS_WINDOW_SIZE) * gridDim.x;
    }
}


/* deflate_state must have been initialized. */
int
culzss_longest_match (deflate_state *s, int size, int is_firstblock, int flush)
{
  hipMemcpyAsync (s->device_in + CULZSS_WINDOW_SIZE,
                   s->host_in + CULZSS_WINDOW_SIZE,
                   size,
                   hipMemcpyHostToDevice);
  checkCudaError ("copy from host_in to device_in");

  lzss_kernel <<< CULZSS_CUDA_NUM_BLOCKS, 1024, 0,
    NULL >>> (s->device_in, s->device_encode, size+CULZSS_WINDOW_SIZE,
                                 is_firstblock);
  checkCudaError ("launch lzss_kernel.");

  hipMemcpyAsync (s->host_encode + CULZSS_WINDOW_SIZE,
                   s->device_encode + CULZSS_WINDOW_SIZE,
                   size,
                   hipMemcpyDeviceToHost, NULL);
  checkCudaError ("Copy from device_on to host_in.");

  hipDeviceSynchronize ();
    int bflush;			/* set if current block must be flushed */

    int cur = 0;
    for (;;)
    {
        if (s->lookahead < MIN_LOOKAHEAD)
        {
            fill_window (s);
            if (s->lookahead < MIN_LOOKAHEAD && flush == Z_NO_FLUSH)
            {
                return need_more;
            }
            if (s->lookahead == 0)
                break;		/* flush the current block */


            if (s->lookahead < MIN_LOOKAHEAD)
            {
                Tracevv ((stderr, "%c", s->window[s->strstart]));
                _tr_tally_lit (s, s->window[s->strstart], bflush);
                s->lookahead--;
                s->strstart++;
                cur++;

                if (bflush)
                    FLUSH_BLOCK (s, 0);
            }
        }

        else
        {
            while (s->lookahead >= MIN_LOOKAHEAD)
            {
                if (cur < CULZSS_WINDOW_SIZE || s->host_encode[cur].dist == 0
                    || s->lookahead - s->host_encode[cur].len <= MIN_LOOKAHEAD)
                {
                    _tr_tally_lit (s, s->window[s->strstart], bflush);
                    s->strstart++;
                    cur++;
                    if (s->lookahead == 0)
                        break;
                    s->lookahead--;
                }
                else
                {
                    int match_length = s->host_encode[cur].len;
                    s->match_length = match_length>s->lookahead? s->lookahead: match_length;
                    _tr_tally_dist (s, s->host_encode[cur].dist,
                                    s->match_length - MIN_MATCH, bflush);
                    s->strstart += s->match_length;
                    cur += s->match_length;
                    s->lookahead -= s->match_length;
                    s->match_length = 0;
                    s->ins_h = s->window[s->strstart];
                }
                if (bflush)
                    FLUSH_BLOCK (s, 0);
            }
        }
    }

    s->insert = s->strstart < MIN_MATCH - 1 ? s->strstart : MIN_MATCH - 1;
    if (flush == Z_FINISH)
    {
        FLUSH_BLOCK (s, 1);
        return finish_done;
    }
    if (s->last_lit)
        FLUSH_BLOCK (s, 0);
    return block_done;
}
